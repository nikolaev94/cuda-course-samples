#include "hip/hip_runtime.h"

#include "cufft_task.hpp"

__global__ void convolution(const complex* a, const complex* b, complex *c, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx].x = a[idx].x * b[idx].x / n - a[idx].y * b[idx].y / n;
        c[idx].y = a[idx].x * b[idx].y / n + a[idx].y * b[idx].x / n;
    }
}

void fft_gpu(const complex* a, const complex* b, complex* c, int n)
{
    hipfftHandle plan;
    hipfftComplex* a_gpu = nullptr, *b_gpu = nullptr, *c_gpu = nullptr;
    hipMalloc((void**)&a_gpu, sizeof(hipfftComplex) * n);
    hipMalloc((void**)&b_gpu, sizeof(hipfftComplex) * n);
    hipMalloc((void**)&c_gpu, sizeof(hipfftComplex) * n);
    hipMemcpy(a_gpu, a, sizeof(hipfftComplex) * n, hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, sizeof(hipfftComplex) * n, hipMemcpyHostToDevice);
    hipMemcpy(c_gpu, c, sizeof(hipfftComplex) * n, hipMemcpyHostToDevice);
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, a_gpu, a_gpu, HIPFFT_FORWARD);
    hipfftExecC2C(plan, b_gpu, b_gpu, HIPFFT_FORWARD);
    unsigned num_blocks = (n + FFT_BLOCK_SIZE - 1) / FFT_BLOCK_SIZE;
    convolution <<<num_blocks, FFT_BLOCK_SIZE>>> (a_gpu, b_gpu, c_gpu, n);
    hipDeviceSynchronize();
    hipfftExecC2C(plan, c_gpu, c_gpu, HIPFFT_BACKWARD);
    hipMemcpy(c, c_gpu, sizeof(hipfftComplex) * n, hipMemcpyDeviceToHost);
    hipfftDestroy(plan);
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);
}
