#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#include "curand_task.hpp"

typedef hiprandStateXORWOW_t randState;

__device__ float test_function(float x)
{
	return (1 / (1 + x * x));
}

__global__ void check_points(float a, float b, unsigned* counts, int n, float* x, float* y)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n)
    {
        float x_norm = a + (b - a) * x[idx];
		if (y[idx] >= test_function(x_norm)) {
			counts[idx] = 0;
		} else {
			counts[idx] = 1;
		}
	}
}

__global__ void setup_random_state(randState* states, unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void generate_check_points(randState* states, float a, float b, unsigned* counts, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        float x_norm = a + (b - a) * hiprand_uniform(&states[idx]);
        float y_gen = hiprand_uniform(&states[idx]);
        if (y_gen >= test_function(x_norm)) {
            counts[idx] = 0;
        }
        else {
            counts[idx] = 1;
        }
    }
}

void montecarlo_gpu_host_api(float a, float b, unsigned* counts , int n)
{
	float* x_gpu = nullptr, *y_gpu = nullptr;
    unsigned* counts_gpu = nullptr;
	hipMalloc((void**)&x_gpu, sizeof(float) * n);
	hipMalloc((void**)&y_gpu, sizeof(float) * n);
	hipMalloc((void**)&counts_gpu, sizeof(unsigned) * n);
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_XORWOW);
	hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));
	hiprandGenerateUniform(generator, x_gpu, n);
	hiprandGenerateUniform(generator, y_gpu, n);
	hiprandDestroyGenerator(generator);
	unsigned num_blocks = (n + MCARLO_BLOCK_SIZE - 1) / MCARLO_BLOCK_SIZE;
	check_points <<< num_blocks, MCARLO_BLOCK_SIZE >>>(a, b, counts_gpu, n, x_gpu, y_gpu);
	hipMemcpy(counts, counts_gpu, sizeof(unsigned) * n, hipMemcpyDeviceToHost);
	hipFree(x_gpu);
	hipFree(y_gpu);
	hipFree(counts_gpu);
}

void montecarlo_gpu_device_api(float a, float b, unsigned* counts, int n) {
	unsigned *counts_gpu = nullptr;
    hiprandStateXORWOW_t* dev_states = nullptr;
    unsigned num_blocks = (n + MCARLO_BLOCK_SIZE - 1) / MCARLO_BLOCK_SIZE;
	hipMalloc((void**)&counts_gpu, sizeof(unsigned) * n);
	hipMalloc((void**)&dev_states, num_blocks * MCARLO_BLOCK_SIZE * sizeof(hiprandStateXORWOW_t));
    setup_random_state <<< num_blocks, MCARLO_BLOCK_SIZE >>> (dev_states, time(nullptr));
    generate_check_points <<< num_blocks, MCARLO_BLOCK_SIZE >>> (dev_states, a, b, counts_gpu, n);
	hipMemcpy(counts, counts_gpu, sizeof(unsigned) * n, hipMemcpyDeviceToHost);
	hipFree(counts_gpu);
	hipFree(dev_states);
}
