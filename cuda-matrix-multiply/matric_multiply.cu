#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "matrix_multiply.hpp"

const unsigned BLOCK_SIZE = 16;

__global__ void mmult(const float* a, const float* b, float* c, float alpha, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx < n && idy < n) {
		c[idy * n + idx] = a[idy * n + idx] + alpha * b[idx * n + idy];
	}
}

void task_gpu(const float* a, const float* b, float* c, float alpha, int n) {
	float* a_gpu = nullptr, *b_gpu = nullptr, *c_gpu = nullptr;
	hipMalloc((void**)&a_gpu, sizeof(float) * n * n);
	hipMalloc((void**)&b_gpu, sizeof(float) * n * n);
	hipMalloc((void**)&c_gpu, sizeof(float) * n * n);
	hipMemcpy(a_gpu, a, n * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_gpu, b, n * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(c_gpu, c, n * n * sizeof(float), hipMemcpyHostToDevice);
    dim3 block_size_3d(BLOCK_SIZE, BLOCK_SIZE), num_blocks_3d;
	num_blocks_3d.x = (n * n + block_size_3d.x - 1) / block_size_3d.x;
	num_blocks_3d.y = (n * n + block_size_3d.y - 1) / block_size_3d.y;
	mmult <<<num_blocks_3d, block_size_3d>>>(a_gpu, b_gpu, c_gpu, alpha, n);
	hipDeviceSynchronize();
	hipMemcpy(c, c_gpu, n * n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(a_gpu);
	hipFree(b_gpu);
	hipFree(c_gpu);
}
