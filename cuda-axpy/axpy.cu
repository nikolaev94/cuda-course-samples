#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>

#include <hip/hip_runtime_api.h>

#include "axpy.hpp"

__global__ void sp_kernel(int n, float a, float* x, int incx, float* y, int incy)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (((idx * incx) < n) && ((idx * incy) < n))
    {
        y[idx * incy] = y[idx * incy] + a * x[idx * incx];;\
    }
}

__global__ void dp_kernel(int n, double a, double* x, int incx, double* y, int incy)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (((idx * incx) < n) && ((idx * incy) < n))
    {
        y[idx * incy] = y[idx * incy] + a * x[idx * incx];
    }
}

void saxpy_gpu(int n, float a, float* x, int incx, float* y, int incy)
{
	float* x_gpu = nullptr, *y_gpu = nullptr;
    (hipMalloc((void**)&x_gpu, n * sizeof(float)) == hipSuccess);
	(hipMalloc((void**)&y_gpu, n * sizeof(float)) == hipSuccess);
	(hipMemcpy(x_gpu, x, n * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    (hipMemcpy(y_gpu, y, n * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
	int num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	sp_kernel <<<num_blocks, BLOCK_SIZE>>>(n, a, x_gpu, incx, y_gpu, incy);
    (hipDeviceSynchronize() == hipSuccess);
	(hipMemcpy(y, y_gpu, n * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);
    (hipFree(x_gpu) == hipSuccess);
	(hipFree(y_gpu) == hipSuccess);
}

void  daxpy_gpu(int n, double a, double* x, int incx, double* y, int incy)
{
	double* x_gpu = nullptr, *y_gpu = nullptr;
	(hipMalloc((void**)&x_gpu, n * sizeof(double)) == hipSuccess);
    (hipMalloc((void**)&y_gpu, n * sizeof(double)) == hipSuccess);
	(hipMemcpy(x_gpu, x, n * sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
	(hipMemcpy(y_gpu, y, n * sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
	int num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dp_kernel <<<num_blocks, BLOCK_SIZE>>>(n, a, x_gpu, incx, y_gpu, incy);
    (hipDeviceSynchronize() == hipSuccess);
    (hipMemcpy(y, y_gpu, n * sizeof(double), hipMemcpyDeviceToHost) == hipSuccess);
    (hipFree(x_gpu) == hipSuccess);
    (hipFree(y_gpu) == hipSuccess);
}
